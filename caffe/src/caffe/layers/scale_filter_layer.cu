#include "hip/hip_runtime.h"
/*
All modification made by Intel Corporation: © 2016 Intel Corporation

All contributions by the University of California:
Copyright (c) 2014, 2015, The Regents of the University of California (Regents)
All rights reserved.

All other contributions:
Copyright (c) 2014, 2015, the respective contributors
All rights reserved.
For the list of contributors go to https://github.com/BVLC/caffe/blob/master/CONTRIBUTORS.md


Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

    * Redistributions of source code must retain the above copyright notice,
      this list of conditions and the following disclaimer.
    * Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.
    * Neither the name of Intel Corporation nor the names of its contributors
      may be used to endorse or promote products derived from this software
      without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <vector>

#include "caffe/layers/scale_filter_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ScaleFilterForward(const int n, const Dtype* in,
    const Dtype* scales, const int dim, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] * scales[index / dim];
  }
}

template <typename Dtype>
void ScaleFilterLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const int count = bottom[0]->count();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();

  if (this->phase_ == TRAIN) {
    const int dim = bottom[0]->shape(2) * bottom[0]->shape(3);
    Dtype* scales = rand_vec_.mutable_gpu_data();
    caffe_gpu_rng_uniform(rand_vec_.count(), min_scale_, max_scale_, scales);

    // NOLINT_NEXT_LINE(whitespace/operators)
    ScaleFilterForward<Dtype><<<CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS>>>(count, bottom_data, scales, dim, top_data);
    CUDA_POST_KERNEL_CHECK;
  } else {
    caffe_copy(count, bottom_data, top_data);
  }
}

template <typename Dtype>
__global__ void ScaleFilterBackward(const int n, const Dtype* in_diff,
    const Dtype* scales, const int dim, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * scales[index / dim];
  }
}

template <typename Dtype>
void ScaleFilterLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const int count = bottom[0]->count();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();

    if (this->phase_ == TRAIN) {
      const int dim = bottom[0]->shape(2) * bottom[0]->shape(3);
      const Dtype* scales = rand_vec_.gpu_data();

      // NOLINT_NEXT_LINE(whitespace/operators)
      ScaleFilterBackward<Dtype><<<CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS>>>(count, top_diff, scales, dim, bottom_diff);
      CUDA_POST_KERNEL_CHECK;
    } else {
      caffe_copy(count, top_diff, bottom_diff);
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(ScaleFilterLayer);

}  // namespace caffe
