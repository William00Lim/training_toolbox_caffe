#include "hip/hip_runtime.h"
/*
All modification made by Intel Corporation: © 2016 Intel Corporation

All contributions by the University of California:
Copyright (c) 2014, 2015, The Regents of the University of California (Regents)
All rights reserved.

All other contributions:
Copyright (c) 2014, 2015, the respective contributors
All rights reserved.
For the list of contributors go to https://github.com/BVLC/caffe/blob/master/CONTRIBUTORS.md


Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

    * Redistributions of source code must retain the above copyright notice,
      this list of conditions and the following disclaimer.
    * Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.
    * Neither the name of Intel Corporation nor the names of its contributors
      may be used to endorse or promote products derived from this software
      without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// ------------------------------------------------------------------
// Fast R-CNN
// copyright (c) 2015 Microsoft
// Licensed under The MIT License [see fast-rcnn/LICENSE for details]
// Written by Ross Girshick
// Modified by Wei Liu
// ------------------------------------------------------------------

#include <vector>

#include "caffe/layers/smooth_L1_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SmoothL1Forward(const int n, const Dtype* in, Dtype* out) {
  // f(x) = 0.5 * x^2    if |x| < 1
  //        |x| - 0.5    otherwise
  CUDA_KERNEL_LOOP(index, n) {
    Dtype val = in[index];
    Dtype abs_val = abs(val);
    if (abs_val < 1) {
      out[index] = 0.5 * val * val;
    } else {
      out[index] = abs_val - 0.5;
    }
  }
}

template <typename Dtype>
void SmoothL1LossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  int count = bottom[0]->count();
  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),
      bottom[1]->gpu_data(),
      diff_.mutable_gpu_data());    // d := b0 - b1
  if (has_weights_) {
    caffe_gpu_mul(
        count,
        bottom[2]->gpu_data(),
        diff_.gpu_data(),
        diff_.mutable_gpu_data());  // d := w * (b0 - b1)
  }
  // NOLINT_NEXT_LINE(whitespace/operators)
  SmoothL1Forward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, diff_.gpu_data(), errors_.mutable_gpu_data());
  CUDA_POST_KERNEL_CHECK;

  Dtype loss;
  caffe_gpu_asum(count, errors_.gpu_data(), &loss);
  top[0]->mutable_cpu_data()[0] = loss / bottom[0]->num();
}

template <typename Dtype>
__global__ void SmoothL1Backward(const int n, const Dtype* in, Dtype* out) {
  // f'(x) = x         if |x| < 1
  //       = sign(x)   otherwise
  CUDA_KERNEL_LOOP(index, n) {
    Dtype val = in[index];
    Dtype abs_val = abs(val);
    if (abs_val < 1) {
      out[index] = val;
    } else {
      out[index] = (Dtype(0) < val) - (val < Dtype(0));
    }
  }
}

template <typename Dtype>
void SmoothL1LossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  int count = diff_.count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  SmoothL1Backward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, diff_.gpu_data(), diff_.mutable_gpu_data());
  CUDA_POST_KERNEL_CHECK;
  for (int i = 0; i < 2; ++i) {
    if (propagate_down[i]) {
      const Dtype sign = (i == 0) ? 1 : -1;
      const Dtype alpha = sign * top[0]->cpu_diff()[0] / bottom[i]->num();
      caffe_gpu_axpby(
          bottom[i]->count(),              // count
          alpha,                           // alpha
          diff_.gpu_data(),                // x
          Dtype(0),                        // beta
          bottom[i]->mutable_gpu_diff());  // y
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SmoothL1LossLayer);

}  // namespace caffe
