#include "hip/hip_runtime.h"
/*
All modification made by Intel Corporation: © 2016 Intel Corporation

All contributions by the University of California:
Copyright (c) 2014, 2015, The Regents of the University of California (Regents)
All rights reserved.

All other contributions:
Copyright (c) 2014, 2015, the respective contributors
All rights reserved.
For the list of contributors go to https://github.com/BVLC/caffe/blob/master/CONTRIBUTORS.md


Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

    * Redistributions of source code must retain the above copyright notice,
      this list of conditions and the following disclaimer.
    * Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.
    * Neither the name of Intel Corporation nor the names of its contributors
      may be used to endorse or promote products derived from this software
      without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <vector>

#include "gtest/gtest.h"

#include "caffe/blob.hpp"
#include "caffe/common.hpp"
#include "caffe/filler.hpp"
#include "caffe/layers/im2col_layer.hpp"
#include "caffe/util/im2col.hpp"

#include "caffe/test/test_caffe_main.hpp"

namespace caffe {

// Forward declare kernel functions
template <typename Dtype>
__global__ void im2col_gpu_kernel(const int n, const Dtype* data_im,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w,
    const int height_col, const int width_col,
    Dtype* data_col);

template <typename Dtype, int num_axes>
__global__ void im2col_nd_gpu_kernel(const int n, const Dtype* data_im,
    const int* im_shape, const int* col_shape,
    const int* kernel_shape, const int* pad, const int* stride,
    const int* dilation, Dtype* data_col);

template <typename Dtype>
class Im2colKernelTest : public GPUDeviceTest<Dtype> {
 protected:
  Im2colKernelTest()
        // big so launches > 1024 threads
      : blob_bottom_(new Blob<Dtype>(5, 500, 15, 15)),
        blob_kernel_shape_(new Blob<int>()),
        blob_stride_(new Blob<int>()),
        blob_pad_(new Blob<int>()),
        blob_dilation_(new Blob<int>()),
        blob_top_(new Blob<Dtype>()),
        blob_top_cpu_(new Blob<Dtype>()) {
    FillerParameter filler_param;
    GaussianFiller<Dtype> filler(filler_param);
    filler.Fill(this->blob_bottom_);
    vector<int> dim_blob_shape(1, 2);
    blob_kernel_shape_->Reshape(dim_blob_shape);
    blob_stride_->Reshape(dim_blob_shape);
    blob_pad_->Reshape(dim_blob_shape);
    blob_dilation_->Reshape(dim_blob_shape);

    height_ = blob_bottom_->height();
    width_ = blob_bottom_->width();
    channels_ = blob_bottom_->channels();
    pad_ = 0;
    stride_ = 2;
    dilation_ = 3;
    kernel_size_ = 3;
    height_col_ = (height_ + 2 * pad_ -
        (dilation_ * (kernel_size_ - 1) + 1)) / stride_ + 1;
    width_col_ = (width_ + 2 * pad_ -
        (dilation_ * (kernel_size_ - 1) + 1)) / stride_ + 1;

    for (int i = 0; i < 2; ++i) {
      blob_kernel_shape_->mutable_cpu_data()[i] = kernel_size_;
      blob_stride_->mutable_cpu_data()[i] = stride_;
      blob_pad_->mutable_cpu_data()[i] = pad_;
      blob_dilation_->mutable_cpu_data()[i] = dilation_;
    }
  }

  virtual ~Im2colKernelTest() {
    delete blob_bottom_;
    delete blob_top_;
    delete blob_top_cpu_;
    delete blob_kernel_shape_;
    delete blob_stride_;
    delete blob_pad_;
    delete blob_dilation_;
  }

  Blob<int>* const blob_kernel_shape_;
  Blob<int>* const blob_stride_;
  Blob<int>* const blob_pad_;
  Blob<int>* const blob_dilation_;
  Blob<Dtype>* const blob_bottom_;
  Blob<Dtype>* const blob_top_;
  Blob<Dtype>* const blob_top_cpu_;
  int height_;
  int width_;
  int channels_;
  int pad_;
  int stride_;
  int dilation_;
  int kernel_size_;
  int height_col_;
  int width_col_;
};

TYPED_TEST_CASE(Im2colKernelTest, TestDtypes);

TYPED_TEST(Im2colKernelTest, Test2D) {
  // Reshape the blobs to correct size for im2col output
  this->blob_top_->Reshape(this->blob_bottom_->num(),
          this->channels_ * this->kernel_size_ * this->kernel_size_,
          this->height_col_,
          this->width_col_);

  this->blob_top_cpu_->Reshape(this->blob_bottom_->num(),
          this->channels_ * this->kernel_size_ * this->kernel_size_,
          this->height_col_,
          this->width_col_);

  const TypeParam* bottom_data = this->blob_bottom_->gpu_data();
  TypeParam* top_data = this->blob_top_->mutable_gpu_data();
  TypeParam* cpu_data = this->blob_top_cpu_->mutable_cpu_data();

  // CPU Version
  for (int n = 0; n < this->blob_bottom_->num(); ++n) {
    im2col_cpu(this->blob_bottom_->cpu_data() + this->blob_bottom_->offset(n),
      this->channels_, this->height_, this->width_,
      this->kernel_size_, this->kernel_size_, this->pad_, this->pad_,
      this->stride_, this->stride_, this->dilation_, this->dilation_,
      cpu_data + this->blob_top_cpu_->offset(n));
  }

  // GPU version
  int num_kernels = this->channels_ * this->height_col_ * this->width_col_;
  int default_grid_dim = CAFFE_GET_BLOCKS(num_kernels);

  // Launch with different grid sizes
  for (int grid_div = 2; grid_div <= 8; grid_div++) {
    for (int n = 0; n < this->blob_bottom_->num(); ++n) {
      int grid_dim = default_grid_dim/grid_div;
      // NOLINT_NEXT_LINE(whitespace/operators)
      im2col_gpu_kernel<TypeParam><<<grid_dim, CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, bottom_data + this->blob_bottom_->offset(n),
        this->height_, this->width_, this->kernel_size_, this->kernel_size_,
        this->pad_, this->pad_, this->stride_, this->stride_,
        this->dilation_, this->dilation_,
        this->height_col_, this->width_col_,
        top_data + this->blob_top_->offset(n));
      CUDA_POST_KERNEL_CHECK;
    }

    // Compare results against CPU version
    for (int i = 0; i < this->blob_top_->count(); ++i) {
      TypeParam cpuval = cpu_data[i];
      TypeParam gpuval = this->blob_top_->cpu_data()[i];
      EXPECT_EQ(cpuval, gpuval);
      if (cpuval != gpuval) {
        break;
      }
    }
  }
}

TYPED_TEST(Im2colKernelTest, TestND) {
  // Reshape the blobs to correct size for im2col output
  this->blob_top_->Reshape(this->blob_bottom_->num(),
      this->channels_ * this->kernel_size_ * this->kernel_size_,
      this->height_col_,
      this->width_col_);

  this->blob_top_cpu_->ReshapeLike(*this->blob_top_);

  const TypeParam* bottom_data_cpu = this->blob_bottom_->cpu_data();
  TypeParam* top_data_cpu = this->blob_top_cpu_->mutable_cpu_data();

  // CPU Version
  for (int n = 0; n < this->blob_bottom_->num(); ++n) {
    im2col_nd_cpu(bottom_data_cpu + this->blob_bottom_->offset(n), 2,
        this->blob_bottom_->shape().data() + 1,
        this->blob_top_cpu_->shape().data() + 1,
        this->blob_kernel_shape_->cpu_data(),
        this->blob_pad_->cpu_data(), this->blob_stride_->cpu_data(),
        this->blob_dilation_->cpu_data(),
        top_data_cpu + this->blob_top_cpu_->offset(n));
  }

  // GPU version
  int num_kernels = this->channels_ * this->height_col_ * this->width_col_;
  int default_grid_dim = CAFFE_GET_BLOCKS(num_kernels);
  const TypeParam* bottom_data_gpu = this->blob_bottom_->gpu_data();

  // Launch with different grid sizes
  for (int grid_div = 2; grid_div <= 8; grid_div++) {
    for (int n = 0; n < this->blob_bottom_->num(); ++n) {
      const int grid_dim = default_grid_dim / grid_div;
      TypeParam* top_data_gpu = this->blob_top_->mutable_gpu_data();
      // NOLINT_NEXT_LINE(whitespace/operators)
      im2col_nd_gpu_kernel<TypeParam, 2><<<grid_dim, CAFFE_CUDA_NUM_THREADS>>>(
          num_kernels, bottom_data_gpu + this->blob_bottom_->offset(n),
          this->blob_bottom_->gpu_shape() + 1, this->blob_top_->gpu_shape() + 1,
          this->blob_kernel_shape_->gpu_data(), this->blob_pad_->gpu_data(),
          this->blob_stride_->gpu_data(), this->blob_dilation_->gpu_data(),
          top_data_gpu + this->blob_top_->offset(n));
      CUDA_POST_KERNEL_CHECK;
    }

    // Compare results against CPU version
    for (int i = 0; i < this->blob_top_->count(); ++i) {
      TypeParam cpuval = top_data_cpu[i];
      TypeParam gpuval = this->blob_top_->cpu_data()[i];
      EXPECT_EQ(cpuval, gpuval);
      if (cpuval != gpuval) {
        break;
      }
    }
  }
}

}  // namespace caffe
